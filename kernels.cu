#include "hip/hip_runtime.h"
__global__ void reduce(float* a, int N) {
    int id = blockIdx.x*blockDim.x + threadIdx.x;
    if (id<N) {
        a[2*id]+=a[2*id+1];
        __syncthreads();
    }
}

__global__ void force_flush (float4 *f, int N) {
    int id = blockIdx.x*blockDim.x + threadIdx.x;
    if (id>=N) return;
    f[id].x=0.;
    f[id].y=0.;
    f[id].z=0.;
}

__global__ void rand_init (hiprandStatePhilox4_32_10_t* states) {
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	hiprand_init(1234, id, 0, &states[id]);
}

__global__ void integrate(float4 *r, float4 *forces, int N, hiprandStatePhilox4_32_10_t* states) {
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	if (id>=N) return;
    
    float4 f=forces[id];
    float4 wn = hiprand_normal4(&states[id]); //Gaussian white noise ~N(0,1)
    //float4 ri=r[id];
    float4 ri=tex1Dfetch(r_t, id);
    float3 dr;
    dr.x=bd_c.hoz*f.x+bd_c.Gamma*wn.x;
    dr.y=bd_c.hoz*f.y+bd_c.Gamma*wn.y;
    dr.z=bd_c.hoz*f.z+bd_c.Gamma*wn.z;
    ri.x+=dr.x;
    ri.y+=dr.y;
    ri.z+=dr.z;
    ri.w=dr.x*dr.x+dr.y*dr.y+dr.z*dr.z;     //Save for calculation of diffusion constant / temperature / kinetic energy
    r[id]=ri;
}

__global__ void minimize(float4 *r, float4 *forces, int N, float alpha) {
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	if (id>=N) return;
    
    float4 f=forces[id];
    //float4 ri=r[id];
    float4 ri=tex1Dfetch(r_t, id);
    ri.x+=alpha*f.x;
    ri.y+=alpha*f.y;
    ri.z+=alpha*f.z;
    r[id]=ri;
}

__global__ void FENEForce(float4* r, float4* forces, InteractionList<bond> list) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i>=list.N) return;
    
    //float4 ri=r[i];
    float4 ri=tex1Dfetch(r_t, i);
    float4 f=forces[i];
    int Nb=list.count_d[i];                 //Number of bonds of the i-th bead
    for (int ib=0; ib<Nb; ib++) {           //Loop over bonds of the i-th bead
        bond b=list.map_d[ib*list.N+i];     //Look up bond in the map
        //float4 l=r[b.i2];                 //Number of bead on the other end of the bond (i2) and its coordinates (l)
        float4 l=tex1Dfetch(r_t, b.i2);     //(reading from texture cache is faster than directly from r[])
        l.x-=ri.x;                          //Atom-to-bead vector
        l.y-=ri.y;
        l.z-=ri.z;
        l.w=sqrtf(l.x*l.x+l.y*l.y+l.z*l.z);
        l.w-=b.l0;
        float denom=(1.-l.w*l.w/fene_c.R02);
        l.w=fene_c.kR0*l.w/denom/(l.w+b.l0);
        f.x+=l.w*l.x;
        f.y+=l.w*l.y;
        f.z+=l.w*l.z;
    }
    forces[i]=f;
}

__global__ void FENEEnergy(float4* r, InteractionList<bond> list) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i>=list.N) return;
    
    //float4 ri=r[i];
    float4 ri=tex1Dfetch(r_t, i);
    float energy=0.0f;
    int Nb=list.count_d[i];                 //Number of bonds of the i-th bead
    for (int ib=0; ib<Nb; ib++) {           //Loop over bonds of the i-th bead
        bond b=list.map_d[ib*list.N+i];     //Look up bond in the map
        //float4 l=r[b.i2];                 //Number of bead on the other end of the bond (i2) and its coordinates (l)
        float4 l=tex1Dfetch(r_t, b.i2);     //(reading from texture cache is faster than directly from r[])
        l.x-=ri.x;                          //Atom-to-bead vector
        l.y-=ri.y;
        l.z-=ri.z;
        l.w=sqrtf(l.x*l.x+l.y*l.y+l.z*l.z);
        l.w-=b.l0;
        l.w=.5*fene_c.k*fene_c.R02*logf(1.-l.w*l.w/fene_c.R02);
        energy+=l.w;
    }
    r[i].w=energy;
}


__global__ void SoftSphereForce(float4 *r, float4 *forces, InteractionList<int> list, float *sig) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i>=list.N) return;
    
    float4 f=forces[i];
    //float4 ri=r[i];
    float4 ri=tex1Dfetch(r_t, i);
    //float sigi=sig[i];
    float sigi=tex1Dfetch(sig_t,i);                                             //Sigma of the i-th bead
    int Nneib=list.count_d[i];                                                  //Number of neighbors of the i-th bead
    for (int ineib=0;ineib<Nneib;ineib++) {                                     //Loop over neighbors of the i-th bead
        int j=list.map_d[ineib*list.N+i];                                       //Look up neibor in the neibor list
        //float4 r2=r[j];
        float4 r2=tex1Dfetch(r_t,j);
        //float4 r2=tex1Dfetch(r_t,tex1Dfetch(neibmap_t,ineib*list.N+i);
        r2.x-=ri.x;
        r2.y-=ri.y;
        r2.z-=ri.z;
        //float sigma2=(sigi+sig[j])/2.;
        float sigma2=(sigi+tex1Dfetch(sig_t,j))/2.;     // sigma of the other bead, and mixed into sigma_ij
        sigma2*=sigma2;
        r2.w=sigma2/(r2.x*r2.x+r2.y*r2.y+r2.z*r2.z);    // squared
        if (r2.w>ss_c.CutOffFactor2inv) {               // Potential is cut off at rcut=CutOffFactor*sigma => sigma^2/r^2 should be > 1/CutOffFactor2
            r2.w*=r2.w;                                 // to the 4th
            r2.w*=r2.w*ss_c.Minus6eps/sigma2;           // to the 8th
            f.x+=r2.x*r2.w;
            f.y+=r2.y*r2.w;
            f.z+=r2.z*r2.w;
        }
    }
    forces[i]=f;
}

__global__ void SoftSphereEnergy(float4 *r, InteractionList<int> list, float *sig) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i>=list.N) return;
    
    //float4 ri=r[i];
    float4 ri=tex1Dfetch(r_t, i);
    float energy=0.0f;
    //float sigi=sig[i];
    float sigi=tex1Dfetch(sig_t,i);                                             //Sigma of the i-th bead
    int Nneib=list.count_d[i];                                                  //Number of neighbors of the i-th bead
    for (int ineib=0;ineib<Nneib;ineib++) {                                     //Loop over neighbors of the i-th bead
        int j=list.map_d[ineib*list.N+i];                                       //Look up neibor in the neibor list
        //float4 r2=r[j];
        float4 r2=tex1Dfetch(r_t,j);
        //float4 r2=tex1Dfetch(r_t,tex1Dfetch(neibmap_t,ineib*list.N+i);
        r2.x-=ri.x;
        r2.y-=ri.y;
        r2.z-=ri.z;
        //float sigma2=(sigi+sig[j])/2.;
        float sigma2=(sigi+tex1Dfetch(sig_t,j))/2.;
        sigma2*=sigma2;
        r2.w=sigma2/(r2.x*r2.x+r2.y*r2.y+r2.z*r2.z); // squared
        if (r2.w>ss_c.CutOffFactor2inv)              // Potential is cut off at rcut=CutOffFactor*sigma => sigma^2/r^2 should be > 1/CutOffFactor2
            energy+=ss_c.eps*r2.w*r2.w*r2.w;         // to the 6th
    }
    r[i].w=energy;
}

__global__ void NativeSubtractSoftSphereForce(float4* r, float4* forces, InteractionList<nc> list, float *sig) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i>=list.N) return;
    
    //float4 ri=r[i];
    float4 ri=tex1Dfetch(r_t, i);
    float4 f=forces[i];
    int Nnc=list.count_d[i];
    //float sigi=sig[i];
    float sigi=tex1Dfetch(sig_t,i);
    for (int inc=0; inc<Nnc; inc++) {
        nc ncij=list.map_d[inc*list.N+i];
        int j=ncij.i2;
        //float4 r2=r[j];
        float4 r2=tex1Dfetch(r_t,j);
        r2.x-=ri.x;
        r2.y-=ri.y;
        r2.z-=ri.z;
        //float sigma2=(sigi+sig[j])/2.;
        float sigma2=(sigi+tex1Dfetch(sig_t,j))/2.;
        sigma2*=sigma2;
        r2.w=sigma2/(r2.x*r2.x+r2.y*r2.y+r2.z*r2.z); // squared
        if (r2.w>ss_c.CutOffFactor2inv) {
            r2.w*=r2.w;                                  // to the 4th
            r2.w*=r2.w*ss_c.Minus6eps/sigma2;           // to the 8th
            f.x-=r2.x*r2.w;
            f.y-=r2.y*r2.w;
            f.z-=r2.z*r2.w;
        }
    }
    forces[i]=f;
}

__global__ void NativeSubtractSoftSphereForce(float4* r, float4* forces, InteractionList<nc> list, float *sig, float Delta) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i>=list.N) return;
    
    //float4 ri=r[i];
    float4 ri=tex1Dfetch(r_t, i);
    float4 f=forces[i];
    int Nnc=list.count_d[i];
    //float sigi=sig[i];
    float sigi=tex1Dfetch(sig_t,i);
    for (int inc=0; inc<Nnc; inc++) {
        nc ncij=list.map_d[inc*list.N+i];
        int j=ncij.i2;
        //float4 r2=r[j];
        float4 r2=tex1Dfetch(r_t,j);
        r2.x-=ri.x;
        r2.y-=ri.y;
        r2.z-=ri.z;
        //float sigma2=(sigi+sig[j])/2.;
        float sigma2=(sigi+tex1Dfetch(sig_t,j))/2.;
        sigma2*=sigma2;
        r2.w=sigma2/(r2.x*r2.x+r2.y*r2.y+r2.z*r2.z); // squared
        if (r2.w>ss_c.CutOffFactor2inv) {
            r2.w*=r2.w;                                  // to the 4th
            r2.w*=Delta*r2.w*ss_c.Minus6eps/sigma2;           // to the 8th
            f.x-=r2.x*r2.w;
            f.y-=r2.y*r2.w;
            f.z-=r2.z*r2.w;
        }
    }
    forces[i]=f;
}

__global__ void NativeSubtractSoftSphereEnergy(float4 *r, InteractionList<nc> list, float *sig) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i>=list.N) return;
    
    //float4 ri=r[i];
    float4 ri=tex1Dfetch(r_t, i);
    float energy=0.0f;
    int Nnc=list.count_d[i];
    //float sigi=sig[i];
    float sigi=tex1Dfetch(sig_t,i);
    for (int inc=0; inc<Nnc; inc++) {
        nc ncij=list.map_d[inc*list.N+i];
        int j=ncij.i2;
        //float4 r2=r[j];
        float4 r2=tex1Dfetch(r_t,j);
        r2.x-=ri.x;
        r2.y-=ri.y;
        r2.z-=ri.z;
        //float sigma2=(sigi+sig[j])/2.;
        float sigma2=(sigi+tex1Dfetch(sig_t,j))/2.;
        sigma2*=sigma2;
        r2.w=sigma2/(r2.x*r2.x+r2.y*r2.y+r2.z*r2.z);      // squared
        if (r2.w>ss_c.CutOffFactor2inv)
            energy-=ss_c.eps*r2.w*r2.w*r2.w;              // to the 6th
    }
    r[i].w=energy;
}

__global__ void NativeForce(float4* r, float4* forces, InteractionList<nc> list) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i>=list.N) return;
    
    //float4 ri=r[i];
    float4 ri=tex1Dfetch(r_t,i);
    float4 f=forces[i];
    int Nnc=list.count_d[i];
    for (int inc=0; inc<Nnc; inc++) {
        nc ncij=list.map_d[inc*list.N+i];
        //float4 r2=r[ncij.i2];
        float4 r2=tex1Dfetch(r_t,ncij.i2);
        r2.x-=ri.x;
        r2.y-=ri.y;
        r2.z-=ri.z;
        r2.w=ncij.r02/(r2.x*r2.x+r2.y*r2.y+r2.z*r2.z);
        float r6inv=r2.w*r2.w*r2.w;
        r2.w=ncij.factor*r2.w*r6inv*(1-r6inv);
        f.x+=r2.w*r2.x;
        f.y+=r2.w*r2.y;
        f.z+=r2.w*r2.z;
    }
    forces[i]=f;
}

__global__ void NativeForce(float4* r, float4* forces, InteractionList<nc> list, float Delta) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i>=list.N) return;
    
    //float4 ri=r[i];
    float4 ri=tex1Dfetch(r_t,i);
    float4 f=forces[i];
    int Nnc=list.count_d[i];
    for (int inc=0; inc<Nnc; inc++) {
        nc ncij=list.map_d[inc*list.N+i];
        //float4 r2=r[ncij.i2];
        float4 r2=tex1Dfetch(r_t,ncij.i2);
        r2.x-=ri.x;
        r2.y-=ri.y;
        r2.z-=ri.z;
        r2.w=ncij.r02/(r2.x*r2.x+r2.y*r2.y+r2.z*r2.z);
        float r6inv=r2.w*r2.w*r2.w;
        r2.w=Delta*ncij.factor*r2.w*r6inv*(1-r6inv);
        f.x+=r2.w*r2.x;
        f.y+=r2.w*r2.y;
        f.z+=r2.w*r2.z;
    }
    forces[i]=f;
}

__global__ void NativeEnergy(float4* r, InteractionList<nc> list) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i>=list.N) return;
    
    //float4 ri=r[i];
    float4 ri=tex1Dfetch(r_t,i);
    float energy=0.0f;
    int Nnc=list.count_d[i];
    for (int inc=0; inc<Nnc; inc++) {
        nc ncij=list.map_d[inc*list.N+i];
        //float4 r2=r[ncij.i2];
        float4 r2=tex1Dfetch(r_t,ncij.i2);
        r2.x-=ri.x;
        r2.y-=ri.y;
        r2.z-=ri.z;
        r2.w=ncij.r02/(r2.x*r2.x+r2.y*r2.y+r2.z*r2.z);
        float r6inv=r2.w*r2.w*r2.w;
        energy+=ncij.epsilon*r6inv*(r6inv-2.0f);
    }
    r[i].w=energy;
}

__global__ void DebyeHuckelForce(float4* r, float4* forces, InteractionList<bond> list) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i>=list.N) return;
    
    //float4 ri=r[i];
    float4 ri=tex1Dfetch(r_t,i);
    float4 f=forces[i];
    int Nsb=list.count_d[i];
    for (int isb=0; isb<Nsb; isb++) {
        bond sbij=list.map_d[isb*list.N+i];
        //float4 r2=r[ncij.i2];
        float4 r2=tex1Dfetch(r_t,sbij.i2);
        r2.x-=ri.x;
        r2.y-=ri.y;
        r2.z-=ri.z;
        float dist2=r2.x*r2.x+r2.y*r2.y+r2.z*r2.z;
        float dist=sqrtf(dist2);
        //if (dist<1.5*els_c.kappainv) {
            r2.w=expf(-dist/els_c.kappainv)*sbij.l0/dist2;
            f.x+=r2.w*r2.x;
            f.y+=r2.w*r2.y;
            f.z+=r2.w*r2.z;
        //}
    }
    forces[i]=f;
}

__global__ void DebyeHuckelEnergy(float4* r, InteractionList<bond> list) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i>=list.N) return;
    
    //float4 ri=r[i];
    float4 ri=tex1Dfetch(r_t,i);
    float energy=0.f;
    int Nsb=list.count_d[i];
    for (int isb=0; isb<Nsb; isb++) {
        bond sbij=list.map_d[isb*list.N+i];
        //float4 r2=r[ncij.i2];
        float4 r2=tex1Dfetch(r_t,sbij.i2);
        r2.x-=ri.x;
        r2.y-=ri.y;
        r2.z-=ri.z;
        float dist2=r2.x*r2.x+r2.y*r2.y+r2.z*r2.z;
        float dist=sqrtf(dist2);
        //if (dist<1.5*els_c.kappainv)
            energy+=expf(-dist/els_c.kappainv)*sbij.l0/dist;
    }
    r[i].w=energy;
}


__global__ void SoftSphereNeighborList(float4* r, InteractionList<int> list) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i>=list.N) return;
    
    //float4 ri=r[i];
    float4 ri=tex1Dfetch(r_t,i);
    int neighbors=0;
    for (int j=0;j<list.N;j++) {
        //float4 r2=r[j];
        float4 r2=tex1Dfetch(r_t,j);
        r2.x-=ri.x;
        r2.y-=ri.y;
        r2.z-=ri.z;
        r2.w=r2.x*r2.x+r2.y*r2.y+r2.z*r2.z;
        if (
            (r2.w<ss_c.Rcut2) and
            (
             (abs(j-i)>1) or
             ((abs(j-i)>0) and ((i>=list.N/2) or (j>=list.N/2)))  //bb with ss or ss with ss on neighboring residues (this actually excludes terminal beads of different chains, that are not bound)
             ) and
            ((j+list.N/2)!=i) and                                 //exclude covalently bonded bb and ss beads
            ((i+list.N/2)!=j)
            ) {
            
            list.map_d[neighbors*list.N+i]=j;
            neighbors++;
        }
    }
    list.count_d[i]=neighbors;
    
}

__global__ void SoftSphereNeighborList(float4* r, InteractionList<int> intlist, InteractionList<int> neiblist) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i>=intlist.N) return;
        
    //float4 ri=r[i];
    float4 ri=tex1Dfetch(r_t,i);
    int Npartners=intlist.count_d[i];
    int neighbors=0;
    for (int ip=0;ip<Npartners;ip++) {
        int j=intlist.map_d[ip*intlist.N+i];
        //float4 r2=r[j];
        float4 r2=tex1Dfetch(r_t,j);
        r2.x-=ri.x;
        r2.y-=ri.y;
        r2.z-=ri.z;
        r2.w=r2.x*r2.x+r2.y*r2.y+r2.z*r2.z;
        if (r2.w<ss_c.Rcut2) {
            neiblist.map_d[neighbors*neiblist.N+i]=j;
            neighbors++;
        }
    }
    neiblist.count_d[i]=neighbors;
}
